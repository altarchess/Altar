#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>




#include "Tinn.h"
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime_api.h>




__global__ void proph(Tinn t, const float* const in) {
    int i = threadIdx.x;
    float sum = 0.0f;
        for (int j = 0; j < t.nips; j++) {
            sum += in[j] * t.w[i * t.nips + j];
        }
        t.h[i] = 1.0f / (1.0f + expf(-(sum + t.b[0])));
}

__global__ 
void propo(Tinn t) {
    int i = threadIdx.x;
    float sum = 0.0f;
       for (int j = 0; j < t.nhid; j++)
           sum += t.h[j] * t.x[i * t.nhid + j];
       t.o[i] = 1.0f / (1.0f + expf(-(sum + t.b[1])));
 
}

Tinn xtbuildgpu (const int nips, const int nhid, const int nops)
{
    Tinn t;
    t.nb = 2;
    t.nw = nhid * (nips + nops);
    hipMallocManaged(&t.w, t.nw * sizeof(*t.w));
    t.x = t.w + nhid * nips;
    hipMallocManaged(&t.b, t.nb * sizeof(*t.b));
    hipMallocManaged(&t.h, nhid * sizeof(*t.h));
    hipMallocManaged(&t.o, nops * sizeof(*t.o));


    t.nips = nips;
    t.nhid = nhid;
    t.nops = nops;
    return t;
}


Tinn xtloadgpu (const char* const path)
{
    FILE* const file = fopen(path, "r");
    int nips = 0;
    int nhid = 0;
    int nops = 0;
    fscanf(file, "%d %d %d\n", &nips, &nhid, &nops);
    const Tinn t = xtbuildgpu(nips, nhid, nops);
    for (int i = 0; i < t.nb; i++) fscanf(file, "%f\n", &t.b[i]);
    for (int i = 0; i < t.nw; i++) fscanf(file, "%f\n", &t.w[i]);
    fclose(file);
    return t;
}
void xtfpropgpu(const Tinn t, const float* const in)
{



    proph<<<1, 64 >>> (t, in);
   // hipDeviceSynchronize();
    propo<<<1, 1>>> (t);
    hipDeviceSynchronize();
}
float* xtpredictgpu(const Tinn t, const float* const in)
{
    xtfpropgpu(t, in);
    return t.o;
}

void xtfreegpu(Tinn t) {
    hipFree(t.w);
    hipFree(t.b);
    hipFree(t.h);
    hipFree(t.o);
}
